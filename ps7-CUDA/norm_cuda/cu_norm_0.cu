#include "hip/hip_runtime.h"
// -*- c++ -*-

#include "Timer.hpp"
#include <iostream>
#include <string>

#include "hip/hip_runtime_api.h"

__global__ 
void dot0(int n, float* a, float* x, float* y) {
  extern __shared__ float sdata[];

  int tid    = threadIdx.x;
  int index  = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  sdata[tid] = 0.0;
  for (int i = index; i < n; i += stride)
    sdata[tid] += x[i] * y[i];

  __syncthreads();

  if (tid == 0) {
    
    a[blockIdx.x] = 0.0;
    for (int i = 0; i < blockDim.x; ++i) {
      a[blockIdx.x] += sdata[i];
    }
  }
}

size_t frob(size_t size) {
  size_t r = 0;
  while(size >>= 1) {
    ++r;
  }
  return 1 << r;
}

size_t find_10ms_size () {
  size_t size = 4;
  Timer t0;

  for (; size < 4UL * 1024 * 1024 * 1024; size *= 2) {
    float *a, *x, *y;
    size_t N = size;

    int block_size = 256;
    int num_blocks = (N + block_size - 1) / block_size;
    int shm_size = block_size * sizeof(float);
  
    hipMallocManaged(&a, num_blocks * sizeof(float));
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));
    t0.start();
    checkCudaErrors(hipDeviceSynchronize());
    dot0<<<num_blocks, block_size, shm_size>>>(N, a, x, y);
    checkCudaErrors(hipDeviceSynchronize());
    t0.stop();
    hipFree(a);
    hipFree(x);
    hipFree(y);
    if (t0.elapsed() >= 10.0) {
      break;
    }
  }

  double rat = std::max<double>(1.0, std::ceil(t0.elapsed()/10.0));
  size /= static_cast<size_t>(rat);

  return frob(size);
}

int main(int argc, char* argv[]) {

  size_t N = 1 << 16;
  int    block_size = 256;

  if (argc >= 2) {
    size_t nn = std::stol(argv[1]);
    N = frob(1 << nn);
  }
  if (argc >= 3) {
    block_size = frob(std::stol(argv[2]));
  }
  int num_blocks = (N + block_size - 1) / block_size;
  int shm_size = block_size * sizeof(float);

  float *a = nullptr, *x = nullptr, *y = nullptr;

  DEF_TIMER(cuda_malloc); START_TIMER(cuda_malloc);
  hipMallocManaged(&a, num_blocks * sizeof(float));
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  STOP_TIMER(cuda_malloc);

  size_t num_trips = 50.0 * find_10ms_size() / static_cast<double>(N);
  num_trips = std::max<size_t>(num_trips, 5);
  
  for (int i = 0; i < N; i++)
    y[i] = 2.0 * (x[i] = 1.0f);
  
  DEF_TIMER(cuda_call);
  START_TIMER(cuda_call);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    dot0<<<num_blocks, block_size, shm_size>>>(N, a, x, y);
    hipDeviceSynchronize();
  }
  STOP_TIMER(cuda_call);

  std::cout << num_blocks << std::endl;

  double ans = 0.0;
  for (int i = 0; i < num_blocks; ++i) {
    ans += a[i];
  }
  std::cout << "ans " << ans << " " << 2.0 * ((double) N) << std::endl;

  double gflops = 2.0 * num_trips * N / 1.e9;
  double gflops_sec = gflops / (t_cuda_call.elapsed() * 1.e-3);
  std::cout << "# gflops / sec [ dot0 ]: " << gflops_sec << std::endl;

  DEF_TIMER(cuda_free); START_TIMER(cuda_free);
  hipFree(a);
  hipFree(x);
  hipFree(y);
  STOP_TIMER(cuda_free);

  return 0;
}
